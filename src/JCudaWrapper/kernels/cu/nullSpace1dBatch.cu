
#include <hip/hip_runtime.h>
#include <cstdio>

// Utility function for swapping two doubles
__device__ void swap(double& a, double& b) {
    double temp = a;
    a = b;
    b = temp;
}

/**
 * Class representing a single row in a column-major matrix.
 */
class Row {
private:
    double* data; /**< Pointer to the beginning of the row's data. */
    int width;    /**< Width of the matrix. */
    int ld;       /**< Leading dimension of the matrix. */

public:
    /**
     * Constructor for the Row class.
     * @param data Pointer to the row data (column-major format).
     * @param width Width of the matrix.
     * @param ld Leading dimension of the matrix.
     */
    __device__ Row(double* data, int width, int ld) : data(data), width(width), ld(ld) {}

    /**
     * Accessor for elements in the row.
     * @param col The column index of the desired element.
     * @return Reference to the element at the given column index.
     */
    __device__ double& operator()(int col) {
        return data[col * ld];
    }

    /**
     * Performs an in-place row subtraction: this_row -= timesOther * other_row.
     * @param other The row to subtract from this row.
     * @param timesOther The scalar multiplier for the other row.
     */
    __device__ void subtract(Row& other, double timesOther) {
        for (int i = 0; i < width; i++)
            (*this)(i) -= timesOther * other(i);
    }

    /**
     * Swaps the content of this row with another row.
     * @param other The row to swap with.
     */
    __device__ void swap(Row& other) {
        for (int i = 0; i < width; i++)
            ::swap((*this)(i), other(i));
    }
};

/**
 * Class representing a matrix in column-major format.
 */
class Matrix {
private:
    double* data; /**< Pointer to the matrix data. */
    int* pivot;   /**< Pointer to the pivot array. */
    int width;    /**< Width of the matrix. */
    int ld;       /**< Leading dimension of the matrix. */

public:
    /**
     * Constructor for the Matrix class.
     * @param data Pointer to the matrix data (column-major format).
     * @param width Width of the matrix.
     * @param ld Leading dimension of the matrix.
     * @param pivot Pointer to the pivot array.
     */
    __device__ Matrix(double* data, int width, int ld, int* pivot) : data(data), width(width), ld(ld), pivot(pivot) {}

    /**
     * Accessor for elements of the matrix.
     * @param row Row index.
     * @param col Column index.
     * @return Reference to the matrix element at the specified row and column.
     */
    __device__ double& operator()(int row, int col) {
        return data[col * ld + row];
    }

    /**
     * Returns a Row object for the specified row index.
     * @param i The row index.
     * @return A Row object representing the specified row.
     */
    __device__ Row row(int i) {
        return Row(data + i, width, ld);
    }

    /**
     * Performs the work to bring the current row to row echelon form.
     * @param row Current row index.
     * @param col Current column index.
     * @param tolerance Tolerance for determining pivot validity.
     * @return True if a valid pivot was found; False otherwise.
     */
    __device__ bool rowEchelonWorkRow(int row, int col, double tolerance) {
        Row r(data + row, width, ld);
        int swapWith = row;

        // Find a valid pivot row
        while (fabs((*this)(swapWith, col)) <= tolerance && swapWith < width) 
            swapWith++;
        
        if (swapWith != row && swapWith < width) {
            Row needsSwap(data + swapWith, width, ld);
            r.swap(needsSwap);
            pivot[row] = swapWith;
        } else if (swapWith == width) {
            return false; // No valid pivot found
        } else {
            pivot[row] = row; // No swap needed
        }

        // Perform row elimination
        double diagonalElement = (*this)(row, col);
        for (int j = row + 1; j < width; j++) {
            double factor = (*this)(j, col) / diagonalElement;
            Row lower(data + j, width, ld);
            lower.subtract(r, factor);
        }
        return true;
    }

    /**
     * Converts the matrix to row echelon form via Gaussian elimination.
     * Updates the pivot index if a row swap occurs.
     * @param tolerance Tolerance for determining pivot validity.
     */
    __device__ void rowEchelon(double tolerance) {
        for (int row = 0, col = 0; row < width && col < width; col++) {
            if (rowEchelonWorkRow(row, col, tolerance)) 
                row++;
        }
    }

    /**
     * Restores the original row order using the pivot array.
     * @param vec Pointer to the vector to be adjusted.
     */
    __device__ void reversePivot(double* vec) {
        for (int i = width - 1; i >= 0; i--) {
            if (pivot[i] != i) {
                Row r1(data + i, width, ld);
                Row r2(data + pivot[i], width, ld);
                r1.swap(r2);
            }
        }
    }
};

/**
 * CUDA kernel to process a batch of matrices, converting each to row echelon form.
 * @param from Pointer to input matrix array (batch of column-major matrices).
 * @param ldFrom Leading dimension of input matrices.
 * @param to Pointer to output array for nullspace computation. The final element of each vector should be 1.
 * @param ldTo Leading dimension of output matrices.
 * @param batchSize Number of matrices in the batch.
 * @param width Width of each square matrix.
 * @param tolerance Tolerance for numerical operations.
 * @param workSpace Workspace for storing pivot information.
 */
extern "C" __global__ void nullSpace1dBatchKernel(double* from, int ldFrom, double* to, int ldTo, int batchSize, int width, double tolerance, int* workSpace) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batchSize) return;

    Matrix mat(from + width * ldFrom * idx, width, ldFrom, workSpace + width * idx);

    // Shared memory for partial pivoting
    extern __shared__ double sharedPivotValues[];
    mat.rowEchelon(tolerance);

    double* eVec = to + ldTo * idx;

    for (int i = width - 2; i >= 0; i--) {
        eVec[i] = 0;
        for (int j = i + 1; j < width; j++) 
            eVec[i] -= eVec[j] * mat(i, j);
        if (mat(i, i) != 0) 
            eVec[i] /= mat(i, i);
    }

    mat.reversePivot(eVec);
}
