
#include <hip/hip_runtime.h>

extern "C" __global__ void cosKernel(double *from, int incFrom, double *to, int incTo, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[idx * incTo] = cos(from[idx * incFrom]); 
    }
}
//nvcc -ptx cos.cu 