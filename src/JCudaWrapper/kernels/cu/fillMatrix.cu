
#include <hip/hip_runtime.h>
//The first value of from us used to fill this matrix.
//n is the size of the matrix, width * height.

extern "C" __global__ void fillMatrixKernel(double *from, int lda, double *to, int height, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[(idx/height)*lda + idx % height] = from[0]; 
    }
}
//nvcc -ptx fillMatrix.cu 