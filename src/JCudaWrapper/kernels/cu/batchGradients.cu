
#include <hip/hip_runtime.h>
/**
 * Represents multi-dimensional indices for a tensor element.
 */
class Indices {
public:
    const int* dim;       ///< Reference to tensor dimensions.
    const int gradient;   ///< Gradient index (0 = X, 1 = Y, 2 = Z).
    const int layer;      ///< Layer index in the tensor.
    const int col;        ///< Column index in the layer.
    const int row;        ///< Row index in the layer.
    const int srcFlatIndex;  ///< Flat memory index.
    const double* data;   ///< Pointer to tensor data.

    /**
     * Constructs Indices from a flat index.
     * @param threadIndex Flat memory index.
     * @param dim Dimensions of the tensor batch.
     * @param data Pointer to tensor data.
     */
    __device__ Indices(int threadIndex, const int* dim, const double* data)
        : dim(dim),
          gradient(threadIndex / dim[6]),
          layer((threadIndex % dim[5]) / dim[4]),
          col((threadIndex % dim[4]) / dim[0]),
          row(threadIndex % dim[0]),
          srcFlatIndex(threadIndex % dim[6]),
          data(data) {}

    /**
     * Computes a shifted index value in the gradient's direction.
     * @param offset Shift value.
     * @return Flat index of the shifted position.
     */
    __device__ double shift(int offset) const {
        int offsetIndex = 0;
        switch (gradient) {
            case 0: offsetIndex = srcFlatIndex + dim[0] * offset; break;
            case 1: offsetIndex = srcFlatIndex + offset; break;
            case 2: offsetIndex = srcFlatIndex + dim[1] * dim[0] * offset; break;
        }
        return data[offsetIndex];
    }

    /**
     * Computes the gradient using a stencil method.
     * @return Computed gradient value.
     */
    __device__ double grad() const {
        int loc, end;
        switch (gradient) {
            case 0: loc = col; end = dim[1]; break;
            case 1: loc = row; end = dim[0]; break;
            case 2: loc = layer; end = dim[2]; break;
        }

        if (end == 1) return 0.0; // Single element case.

        if (loc == 0) return shift(1) - data[srcFlatIndex]; // Forward difference at start.
        if (loc == end - 1) return data[srcFlatIndex] - shift(-1); // Backward difference at end.
        if (loc == 1 || loc == end - 2) return (shift(1) - shift(-1)) / 2.0; // Central difference.
        return (shift(-2) - 8.0*shift(-1) + 8.0*shift(1) - shift(2))/12.0; // Higher-order stencil.
    }

    /**
     * Prints the current state of the indices for debugging.
     */
    __device__ void print() const {
        printf("srcFlatIndex: %d | Gradient: %d | Layer: %d | Col: %d | Row: %d | Dimensions [H: %d, W: %d, D: %d, N: %d]\n",
               srcFlatIndex, gradient, layer, col, row,
               dim[0], dim[1], dim[2], dim[3]);
    }
};

/**
 * Kernel to compute gradients for batched tensors.
 * @param n Total number of elements in the gradients.
 * @param mat Pointer to input tensor data.
 * @param dim indices height -> 0, width -> 1, depth -> 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
 * @param dX Output gradients along X direction.
 * @param dY Output gradients along Y direction.
 * @param dZ Output gradients along Z direction.
 */
extern "C" __global__ void batchGradientsKernel(
    const int n, 
    const double* mat, 
    const int* dim, //height = 0, width = 1, depth = 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
    double* dX, const int ldx, double* dY, const int ldy, double* dZ, const int ldz
) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (idx >= n) return;

    const Indices indices(idx, dim, mat);

    int dInd = idx%dim[6];
     
    if(idx < dim[6]) dX[(dInd/dim[0])*ldx + dInd%dim[0]] = indices.grad();
    else if(idx < 2*dim[6]) dY[(dInd/dim[0])*ldx + dInd%dim[0]] = indices.grad();
    else dZ[(dInd/dim[0])*ldx + dInd%dim[0]] = indices.grad();
}

