
#include <hip/hip_runtime.h>
/**
 * Represents multi-dimensional indices for a tensor element.
 */
class Indices {
public:
    const int* dim;       ///< Reference to tensor dimensions.
    const int gradient;   ///< Gradient index (0 = X, 1 = Y, 2 = Z).
    const int layer;      ///< Layer index in the tensor.
    const int col;        ///< Column index in the layer.
    const int row;        ///< Row index in the layer.
    const int flatIndex;  ///< Flat memory index.
    const double* data;   ///< Pointer to tensor data.

    /**
     * Constructs Indices from a flat index.
     * @param threadIndex Flat memory index.
     * @param dim Dimensions of the tensor batch.
     * @param data Pointer to tensor data.
     */
    __device__ Indices(int threadIndex, const int* dim, const double* data)
        : dim(dim),
          gradient(threadIndex / dim[6]),
          layer((threadIndex % dim[5]) / dim[4]),
          col((threadIndex % dim[4]) / dim[0]),
          row(threadIndex % dim[0]),
          flatIndex(threadIndex % dim[6]),
          data(data) {}

    /**
     * Computes a shifted index value in the gradient's direction.
     * @param offset Shift value.
     * @return Flat index of the shifted position.
     */
    __device__ double shift(int offset) const {
        int offsetIndex = 0;
        switch (gradient) {
            case 0: offsetIndex = flatIndex + dim[0] * offset; break;
            case 1: offsetIndex = flatIndex + offset; break;
            case 2: offsetIndex = flatIndex + dim[1] * dim[0] * offset; break;
        }
        return data[offsetIndex];
    }

    /**
     * Computes the gradient using a stencil method.
     * @return Computed gradient value.
     */
    __device__ double grad() const {
        int loc, end;
        switch (gradient) {
            case 0: loc = col; end = dim[1]; break;
            case 1: loc = row; end = dim[0]; break;
            case 2: loc = layer; end = dim[2]; break;
        }

        if (end == 1) return 0.0; // Single element case.

        if (loc == 0) return shift(1) - data[flatIndex]; // Forward difference at start.
        if (loc == end - 1) return data[flatIndex] - shift(-1); // Backward difference at end.
        if (loc == 1 || loc == end - 2) return (shift(1) - shift(-1)) / 2.0; // Central difference.
        return (shift(-2) - 8.0*shift(-1) + 8.0*shift(1) - shift(2))/12.0; // Higher-order stencil.
    }

    /**
     * Prints the current state of the indices for debugging.
     */
    __device__ void print() const {
        printf("FlatIndex: %d | Gradient: %d | Layer: %d | Col: %d | Row: %d | Dimensions [H: %d, W: %d, D: %d, N: %d]\n",
               flatIndex, gradient, layer, col, row,
               dim[0], dim[1], dim[2], dim[3]);
    }
};

/**
 * Kernel to compute gradients for batched tensors.
 * @param n Total number of elements in the batch.
 * @param mat Pointer to input tensor data.
 * @param dim indices height -> 0, width -> 1, depth -> 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
 * @param dX Output gradients along X direction.
 * @param dY Output gradients along Y direction.
 * @param dZ Output gradients along Z direction.
 */
extern "C" __global__ void batchGradientsKernel(
    const int n, 
    const double* mat, 
    const int* dim, //height = 0, width = 1, depth = 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
    double* dX, double* dY, double* dZ
) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (idx >= n) return;

    const Indices indices(idx, dim, mat);

    // Compute gradient and store in the corresponding output array.
    double gradValue = indices.grad();

    switch (indices.gradient) {
        case 0: dX[idx] = gradValue; break;
        case 1: dY[idx % dim[6]] = gradValue; break;
        case 2: dZ[idx % dim[6]] = gradValue; break;
    }
}

