
#include <hip/hip_runtime.h>
/**
 * Represents multi-dimensional indices for a tensor element.
 */
class Indices {
public:
    const int* dim;       ///< Reference to tensor dimensions.
    const int gradient;   ///< Gradient index (0 = X, 1 = Y, 2 = Z).
    const int srcFlatIndex;  ///< Flat memory index.
    const double* data;   ///< Pointer to tensor data.
    const int idx;

    /**
     * Constructs Indices from a flat index.
     * @param threadIndex Flat memory index.
     * @param dim Dimensions of the tensor batch.
     * @param data Pointer to tensor data.
     */
    __device__ Indices(int idx, const int* dim, const double* data)
        : dim(dim),
          gradient(idx / dim[6]),
          srcFlatIndex(((idx % dim[6]) / dim[0]) * dim[7] + idx % dim[0]),
          data(data),
          idx(idx) {}

    /**
     * Computes a shifted index value in the gradient's direction.
     * @param offset Shift value.
     * @return Flat index of the shifted position.
     */
    __device__ double shift(int offset) const {
        int offsetIndex;
        switch (gradient) {
            case 0: offsetIndex = srcFlatIndex + dim[7] * offset; break;
            case 1: offsetIndex = srcFlatIndex + offset; break;
            case 2: offsetIndex = srcFlatIndex + dim[1] * dim[7] * offset; break;
        }
        return data[offsetIndex];
    }

    /**
     * Computes the gradient using a stencil method.
     * @return Computed gradient value.
     */
    __device__ double grad() const {
        int loc, end;
        switch (gradient) {
            case 0: loc = (idx % dim[4]) / dim[0]; end = dim[1]; break;
            case 1: loc = idx % dim[0];            end = dim[0]; break;
            case 2: loc = (idx % dim[5]) / dim[4]; end = dim[2]; break;
        }


//		if(idx == 161) printf("id = %d with gradient id %d, has layer %d \n", idx, gradient, (idx % dim[5]) / dim[4]);
//		if(idx >= 162) printf("id = %d with gradient id %d, has layer %d, note: tensor size = %d  and layer size = %d\n", idx, gradient, loc, dim[5], dim[4]);
	//	if(idx == 171) printf("id = %d with gradient id %d, has layer %d \n", idx, gradient, loc);

        if (end == 1) return 0.0; // Single element case.
        if (loc == 0) return shift(1) - data[srcFlatIndex]; // Forward difference at start.
        if (loc == end - 1) return data[srcFlatIndex] - shift(-1); // Backward difference at end.
        if (loc == 1 || loc == end - 2) return (shift(1) - shift(-1)) / 2.0; // Central difference.
        return (shift(-2) - 8.0*shift(-1) + 8.0*shift(1) - shift(2))/12.0; // Higher-order stencil.
    }

};

/**
 *Computes the indices written to.
 */
class DstIndices{
private:
	const int col;
	const int row;
public:
	/**
	 * @param height the height of the matrix.
	 * @param batchSize the number of elements in the batch.
	 * @param idx the thread id.
	 */
	__device__ DstIndices(int height, int batchSize, int idx): row(idx%height), col((idx % batchSize)/height){}
	/**
	 * Computes the index on the destination matrix.
	 * @param ldDst the leading dimension of the destination for which the index is computed.
	 * @param dInd the index in the batch (idx % batchSize).
	 * @param height the height of the matrix.
	 */
	__device__ int index(int ldDst) const {
		return col*ldDst + row;
	}
};
/**
 * Kernel to compute gradients for batched tensors.
 * @param n Total number of elements in the gradients.
 * @param mat Pointer to input tensor data.
 * @param dim indices height -> 0, width -> 1, depth -> 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
 * @param dX Output gradients along X direction.
 * @param dY Output gradients along Y direction.
 * @param dZ Output gradients along Z direction.
 */
extern "C" __global__ void batchGradientsKernel(
    const int n, 
    const double* mat, 
    const int* dim, //height = 0, width = 1, depth = 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6, ld = 7
    double* dX, const int ldx, double* dY, const int ldy, double* dZ, const int ldz
) {    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (idx >= n) return;

    const Indices indices(idx, dim, mat);

    const DstIndices to(dim[0], dim[6], idx);
     
    if(idx < dim[6])        dX[to.index(ldx)] = indices.grad();
    else if(idx < 2*dim[6]) dY[to.index(ldy)] = indices.grad();
    else                    dZ[to.index(ldz)] = indices.grad();
}

