/**
 * @file eigenVecBatchKernel.cu
 * @brief CUDA kernel for computing eigenvectors using row echelon form.
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

/**
 * @class Get
 * @brief A helper class for accessing values in a column-major order matrix.
 */
class Get{
private:
    const int height;
    const int idx;
public:
    __device__ Get(const int idx, const int height): idx(idx), height(height){}
    
    /**
     * @brief Retrieves a value from a column-major order matrix.
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The value at the corresponding column-major index.
     */
    __device__ double val(const double* src, const int  ld) const{
	return val(src, height, ld);
    }
    
    /**
     * @brief Retrieves a value from a column-major order matrix.
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @param height The height of the matrix.
     * @return The value at the corresponding column-major index.
     */
    __device__ double val(const double* src, const int height, const int ld) const{
	return src[ind(height, ld)];
    }
    
    /**
     * @brief Retrieves an index from a column-major order matrix.
     * @param height The height of the matrix.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The computed column-major index.
     */
    __device__ int ind(const int height, const int ld) const{
	return (idx / height) * ld + (idx % height);
    }
    
    /**
     * @brief Retrieves an index from a column-major order matrix using stored height.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The computed column-major index.
     */
    __device__ int ind(const int ld) const{
	return ind(height, ld);
    }
    
    
};


/**
 * @brief Utility function to swap two double values.
 * @param a Reference to the first double.
 * @param b Reference to the second double.
 */
__device__ void swap(double& a, double& b) {
    double temp = a;
    a = b;
    b = temp;
}


/**
 * @class MaxAbs
 * @brief A utility class for tracking the argument corresponding to the maximum absolute value in a set of comparisons.
 *
 * This class is designed for use in CUDA device code and provides methods to update the tracked maximum 
 * absolute value and retrieve the corresponding argument.
 */
class MaxAbs {
private:
    int arg; ///< The argument corresponding to the maximum absolute value.
    double val; ///< The maximum absolute value encountered so far.    

public:
    /**
     * @brief Constructor for the MaxAbs class.
     * 
     * Initializes the maximum absolute value and its corresponding argument.
     *
     * @param initVal The initial maximum absolute value.
     * @param initArg The initial argument corresponding to the maximum absolute value.
     */
    __device__ MaxAbs(int initArg, double initVal) : arg(initArg), val(initVal) {}

     /**
     * @brief Updates the tracked maximum absolute value if the new value is greater.
     * 
     * Compares the given value with the current maximum absolute value. If the new value is greater,
     * updates the maximum value and its corresponding index.
     *
     * @param candidateIndex The index associated with the new value.
     * @param candidateValue The new value to compare against the current maximum absolute value.
     */
    __device__ void challenge(int candidateIndex, double candidateValue) {
        double absoluteValue = fabs(candidateValue); // Compute the absolute value of the candidate value.
        if (absoluteValue > val) {             // Update if the candidate value is larger than the current maximum.
            val = absoluteValue;
            arg = candidateIndex;
        }
    }

    /**
     * @brief Retrieves the argument corresponding to the maximum absolute value.
     *
     * @return The argument corresponding to the maximum absolute value.
     */
    __device__ int getArg() {
        return arg;
    }
    
    /**
     * @brief Retrieves the absolute value at the argument.
     *
     * @return The maximum absolute value.
     */
    __device__ double getVal() {
        return val;
    }
};


/**
 * @class Matrix
 * @brief Represents a matrix and provides utility functions for matrix operations. * 
 */
class Matrix {
private:
    double mat[3][3];     
    int* isPivot; ///< Pointer to an array indicating pivot columns.
    const double tolerance;
    

public:
    /**
     * @brief Constructor for Matrix.
     * @param xx, xy, xz, yy, yz, zz Matrix elements.
     * @param eigenVal Eigenvalue for computation.
     * @param isPivot Pointer to pivot flag array.
     * @param tolerance Numerical tolerance for pivot detection.
     */
    __device__ Matrix(double xx, double xy, double xz, double yy, double yz, double zz, double eigenVal, int* isPivot, double tolerance) 
    : isPivot(isPivot), tolerance(tolerance) {
        mat[0][0] = xx - eigenVal; mat[0][1] = xy; mat[0][2] = xz;
        mat[1][0] = xy; mat[1][1] = yy - eigenVal; mat[1][2] = yz;
        mat[2][0] = xz; mat[2][1] = yz; mat[2][2] = zz - eigenVal;
    }


    /**
     * @brief Access an element in the matrix by row and column index.
     * @param row Row index.
     * @param col Column index.
     * @return Reference to the element at the specified row and column.
     */
    __device__ double& operator()(int row, int col) {
        return mat[row][col];
    }
    
    /**
     * @brief Subtracts a scaled row from another row.
     * @param minuendInd Index of the row to be updated.
     * @param subtrahendInd Index of the row to subtract.
     * @param scale Scaling factor.
     */
    __device__ void subtractRow(int minuendInd, int subtrahendInd, double scale) {
        for (int i = 0; i < 3; i++) mat[minuendInd][i] -= scale * mat[i][subtrahendInd];        
    }

    /**
     * @brief Swaps two rows of the matrix.
     * @param i First row index.
     * @param j Second row index.
     */
    __device__ void swapRows(int i, int j) {
        for(int k = 0; k < 3; k++) swap(mat[i][k], mat[j][k]);
    }

    
    /**
     * @brief Perform row echelon work for a specific row and column.
     * @param row Current row index.
     * @param col Current column index.
     * @return True if a pivot was found, false otherwise.
     */
    __device__ bool reduceToRowEchelon(int row, int col) {
        
        MaxAbs maxPivot(row, fabs(mat[row][col]));
        
	for (int i = row + 1; i < 3; i++) maxPivot.challenge(i, mat[i][col]);

        if (maxPivot.getVal() <= tolerance) return false;

        if (maxPivot.getArg() != row) swapRows(maxPivot.getArg(), row);
        
        for (int i = row + 1; i < 3; i++)
	    subtractRow(i, row, mat[i][col]/mat[row][col]);

        return true;
    }

    /**
     * @brief Perform row echelon reduction on the matrix.
     * @return Number of free variables found during the reduction.
     */
    __device__ int rowEchelon() {
        int numFreeVariables = 0;
        int row = 0;

        for (int col = 0; col < 3; col++) {            
            if (reduceToRowEchelon(row, col)) {
                row++;
                isPivot[col] = 1;
            } else {
                isPivot[col] = 0;
                numFreeVariables++;
            }
        }

        if (fabs(mat[0][0]) < tolerance) isPivot[0] = 0;

        return numFreeVariables;
    }

};


/**
 * This method should be called on a fresh copy of the matrices for which the vectors are sought for each eigenvalue.  Each time with an incremented value of valIndex.
 *
 * @brief CUDA kernel to compute eigenvectors in batch using row echelon form.
 * @param batchSize Number of matrices.
 * @param src Pointer to source matrices in column-major format.  These matrices will be changed.
 * @param ldsrc Leading dimension of the source matrices.
 * @param eVectors Pointer to the resulting eigenvectors.
 * @param width Number of columns in each matrix.
 * @param eigenValues Pointer to all the eigenValues, including those that will not be used.  Be sure to increment valIndex over multiple runs of this kernel so that they are all used.
 * @param workspacePivotFlags Pointer to workspace memory for pivot flags.
 * @param tolerance Tolerance for row echelon pivot detection.
 * @param ldEVec the leading dimension of the eigen vectors.
 * @param ldSrc the leading dimension of the sourver matrix.
 * @param valIndex The index of the desired eigen value. 
 */
extern "C" __global__ void eigenVecBatch3x3Kernel(
     const int batchSize, 
     const double* xx, const int ldxx, 
     const double* xy, const int ldxy, 
     const double* xz, const int ldxz,
     const double* yy, const int ldyy,
     const double* yz, const int ldyz,
     const double* zz, const int ldzz, 
     const int srcHeight, 
    
     double* eVectors,
     const int ldEVec,
     const int heightEVec,
         
     const double* eigenValues,
     const int ldEVal,
     const int heightEVal,
     
     int* workspacePivotFlags,
     const int ldPivot,
     const int heightPivot, 
     
     
     const double tolerance
) {    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batchSize) return;
    
    Get getx3(3*idx, heightEVal);
    
    int* isPivot = workspacePivotFlags + getx3.ind(heightPivot, ldPivot);    
    
    double eigenVal = getx3.val(eigenValues, ldEVal);    
    
    Get get(idx, srcHeight);
    Matrix mat(
        get.val(xx, ldxx), 
        get.val(xy, ldxy), 
        get.val(xz, ldxz), 
        get.val(yy, ldyy), 
        get.val(yz, ldyz), 
        get.val(zz, ldzz),
        eigenVal,
        isPivot, 
	tolerance
    );
    
    double* eVec = eVectors + getx3.ind(heightEVec, ldEVec);
    int numFreeVariables = mat.rowEchelon();

    int col = 2;
    
    while(isPivot[col]) {
    	eVec[col] = 0;
    	col--;
    }
    
    eVec[col] = 1;
    
    for (int row = col - 1; row >= 0 && col >= 0; col--) {	
    	eVec[col] = 0;	
	if(isPivot[col]){         
            for (int i = col + 1; i < 3; i++) 
                eVec[col] -= eVec[i] * mat(row, i);
            eVec[col] /= mat(row, col);
            row--;
        }
    }    
}
