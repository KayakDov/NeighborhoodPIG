
#include <hip/hip_runtime.h>
/**
 * Represents a position in a 2D or 3D matrix and provides utility methods for neighborhood operations.
 * The class abstracts pixel movement along a defined direction (row, column, or depth) and allows 
 * efficient access to source and target matrices during CUDA kernel execution.
 */
class Pixel {
private:
    double* sourceMat;   /**< Pointer to the source matrix. */
    double* targetMat;   /**< Pointer to the target matrix. */
    int stepSize;        /**< Step size for moving along the specified direction. */
    int toInc;           /**< Increment multiplier for accessing the target matrix. */

public:
    /**
     * Constructs a Pixel instance.
     *
     * @param sourceMat Pointer to the source matrix.
     * @param targetMat Pointer to the target matrix.
     * @param stepSize The step size for moving along a direction in the matrix.
     * @param toInc The stride multiplier for accessing the target matrix.
     */
    __device__ Pixel(double* sourceMat, double* targetMat, int stepSize, int toInc)
        : sourceMat(sourceMat), targetMat(targetMat), stepSize(stepSize), toInc(toInc) {}

    /**
     * Retrieves the value from the source matrix at the current position plus an offset.
     *
     * @param offset The offset to apply (in units of `stepSize`).
     * @return The value from the source matrix.
     */
    __device__ double sourceValue(int offset = 0) const {
        return sourceMat[offset * stepSize];
    }

    /**
     * Accesses the value in the target matrix at the current position plus an offset.
     *
     * @param offset The offset to apply (in units of `stepSize`).
     * @return Reference to the value in the target matrix.
     */
    __device__ double& targetValue(int offset = 0) {
        return targetMat[offset * stepSize * toInc];
    }

    /**
     * Advances the pixel to the next position along the direction.
     */
    __device__ void move() {
        sourceMat += stepSize;
        targetMat += stepSize * toInc;
    }
};

/**
 * CUDA kernel for computing a rolling neighborhood sum over a 2D or 3D matrix along a specified direction.
 *
 * @param sourceMat Pointer to the source matrix in global memory.
 * @param targetMat Pointer to the target matrix in global memory.
 * @param height The height of the matrix.
 * @param width The width of the matrix.
 * @param depth The depth of the matrix (3rd dimension).
 * @param toInc The stride multiplier for accessing the target matrix.
 * @param neighborhoodSize The size of the neighborhood window for summation.
 * @param dir Direction of operation: 0 (row), 1 (column), or 2 (depth).
 */
extern "C" __global__ void neighborhoodSum3dKernel(
    int n,
    double* sourceMat,
    double* targetMat,
    int height, int width, int depth,
    int toInc,
    int neighborhoodSize,
    int dir
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return; // Out-of-bounds thread

    // Initialize starting position and step sizes
    int startIdx, stepSize, numSteps;
    switch (dir) {
        case 0: // Row-wise
            startIdx = (idx % height) + (idx / height) * (height * width);
            stepSize = height;
            numSteps = width;
            break;
        case 1: // Column-wise
            startIdx = idx * height;
            stepSize = 1;
            numSteps = height;
            break;
        case 2: // Depth-wise
            startIdx = idx;
            stepSize = height * width;
            numSteps = depth;
    }

    Pixel pixel(sourceMat + startIdx, targetMat + startIdx * toInc, stepSize, toInc);

    int rollingSum = 0;
    for (int i = 0; i <= neighborhoodSize; i++)
        rollingSum += pixel.sourceValue(i);
    
    pixel.targetValue() = rollingSum;

    int i = 1;
    for (; i < numSteps - neighborhoodSize; i++) {
        pixel.move();    
        rollingSum += pixel.sourceValue(neighborhoodSize) - pixel.sourceValue(-neighborhoodSize - 1);
        pixel.targetValue() = rollingSum;
    }

    // Compute the rolling sum for the final region
    for (; i < numSteps; i++) {
        pixel.move();
        rollingSum -= pixel.sourceValue(-neighborhoodSize - 1);
        pixel.targetValue() = rollingSum;
    }
}

