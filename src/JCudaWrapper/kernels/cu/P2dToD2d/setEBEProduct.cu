
#include <hip/hip_runtime.h>
/**
 * @class Get
 * @brief A helper class for accessing and modifying values in a batch of 4D data 
 *        (frames × depth × height × width), where each 2D slice (height × width) 
 *        is stored in column-major order.
 *
 * This class computes the appropriate index to retrieve or update a value from 
 * the flattened memory representation using a linear index. It supports accessing 
 * multi-frame volumetric data with strided memory layout.
 */
class Get {
private:
    const int height;     ///< Height of each 2D slice.
    const int layerSize;
    const int idx;        ///< Linear index of the element processed by the thread.
    
    const int layer;      ///< Index along the depth dimension (z-axis) within a frame.
    const int frame;      ///< Index of the frame in the 4D dataset (time dimension).
    
    const int col;
    const int row;
    
public:
    /**
     * @brief Constructs a Get object to calculate indices for accessing a 4D data batch.
     * 
     * @param inputIdx The linear index of the current thread.
     * @param height   The height (number of rows) of each 2D slice.
     * @param width    The width (number of columns) of each 2D slice.
     * @param depth    The number of slices along the depth (z) dimension per frame.
     */
    __device__ Get(const int idx, const int* dim)
    : idx(idx), 
      height(dim[0]),
      layerSize(dim[4]),
      
      layer((idx % dim[5]) / dim[4]), 
      frame(idx / dim[5]),
      
      col((idx % dim[4]) / dim[0]),
      row(idx % dim[0]) {
      
//      printf("idx = %d, layer = %d, (idx %% dim[5]) = %d, (idx %% dim[5]) / layerSize = %d\n", idx, layer, (idx % dim[5]), (idx % dim[5]) / layerSize);
      
      }

    /**
     * @brief Retrieves a value from the source 4D dataset using the calculated indices.
     *
     * @param src   Array of pointers to 2D slices, arranged in frame-major and then depth-major order.
     * @param ld    Array of leading dimensions for each slice (used for column-major indexing).
     * @param ldld  Leading dimension of the ld array (stride across layers).
     * @param ldPtr Leading dimension of the src array (stride across frames).
     * @return      The double value at the resolved position in the 4D dataset.
     */
    __device__ double operator()(const float** src, const int* ld, const int ldld, const int ldPtr) {
        return src[page(ldPtr)][word(ld, ldld)];
    }
    
    /**
     * @brief Retrieves a value from the source 4D dataset using the calculated indices.
     *
     * @param src   Array of pointers to 2D slices, arranged in frame-major and then depth-major order.
     * @param ld    Array of leading dimensions for each slice (used for column-major indexing).
     * @param ldld  Leading dimension of the ld array (stride across layers).
     * @param ldPtr Leading dimension of the src array (stride across frames).
     * @return      The double value at the resolved position in the 4D dataset.
     */
    __device__ double operator()(double** src, const int* ld, const int ldld, const int ldPtr) {
	return src[page(ldPtr)][word(ld, ldld)];
    }

    /**
     * @brief Sets a value in the destination 4D dataset using the calculated indices.
     *
     * @param src   Array of pointers to 2D slices, arranged in frame-major and then depth-major order.
     * @param ld    Array of leading dimensions for each slice (used for column-major indexing).
     * @param ldld  Leading dimension of the ld array (stride across layers).
     * @param ldPtr Leading dimension of the src array (stride across frames).
     * @param val   The double value to store in the specified location.
     */
    __device__ void set(double** src, const int* ld, const int ldld, const int ldPtr, double val) {
        src[page(ldPtr)][word(ld, ldld)] = val;
    }

    /**
     * @brief Computes the column-major index within the current 2D slice.
     * 
     * @param ld   Array of leading dimensions (strides) for each 2D slice.
     * @param ldld Leading dimension (stride) of the ld array across frames.
     * @return     The resolved column-major index for the current linear thread index.
     */
    __device__ int word(const int* ld, const int ldld) const {
        return col * ld[page(ldld)] + row;
    }

    /**
     * @brief Computes the index into the `src` pointer array to locate the correct 2D slice.
     * 
     * @param ldPtr The leading dimension of the pointer array (`src`), i.e., the number of slices per frame.
     * @return      The index in `src` pointing to the appropriate 2D slice.
     */
    __device__ int page(const int ldPtr) const {
        return frame * ldPtr + layer;
    }
    
    /**
     * @brief Prints the internal state of the Get object and the dim array (for debugging on the host).
     *
     * @param dim The array containing the dimensions of the 4D data.
     */
    __host__ __device__ void print(const int* dim) const {
        printf("Get(idx: %d, frame: %d, layer: %d, height: %d, layerSize: %d, col: %d, row: %d), dim: [%d, %d, %d, %d, %d, %d, %d]\n",
               idx, frame, layer, height, layerSize, col, row,
               dim[0], dim[1], dim[2], dim[3], dim[4], dim[5], (dim + 6)[0]); // Added dim print
    }
};

/**
 * @brief CUDA kernel that performs elementwise computation:
 *        `dst = timesDst * dst + timesProduct * a * b`
 *
 * This kernel operates on 4D batched data (frames × depth × height × width), allowing
 * for strided memory access via pointer arrays and leading-dimension arrays. It reads
 * corresponding values from two inputs (`a` and `b`), multiplies them, scales the result,
 * and accumulates it with a scaled value from the destination (`dst`).
 *
 * @param n         Total number of elements to process (threads).
 * @param dst       Pointer array to output 2D slices (modifiable).
 * @param xyLdDst   Leading dimension array for `dst` (per slice).
 * @param ldldDst   Stride across `xyLdDst` for indexing slices.
 * @param ztLdDst   Stride across `dst` for frame × depth indexing.
 *
 * @param a         Pointer array to 2D slices of input A.
 * @param xyLdA     Leading dimension array for input A slices.
 * @param ldldA     Stride across `xyLdA` for indexing.
 * @param ztLdA     Stride across `a` for frame × depth indexing.
 * 
 * @param b         Pointer to 1D flattened array of input B.
 * @param xyLdB     Leading dimension array for input B.
 * @param ldldB     Stride across `xyLdB` for indexing.
 * @param ztLdB     Stride across `b` for frame × depth indexing.
 * 
 * @param dim       height = 0, width = 1, depth = 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
 * 
 * @param timesProduct Scalar multiplier for the product of `a` and `b`.
 * 
 * 
 * @param timesDst  Scalar multiplier applied to the existing value in `dst`.
 */
extern "C" __global__ void setEBEProductKernel(
    const int n,
    double** dst, const int* xyLdDst, const int ldldDst, const int ztLdDst,
    const float** a, const int* xyLdA, const int ldldA, const int ztLdA,
    const float** b, const int* xyLdB, const int ldldB, const int ztLdB,
    
    const int* dim //height = 0, width = 1, depth = 2, numTensors = 3, layerSize = 4, tensorSize = 5, batchSize = 6
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    Get ind(idx, dim);

//    ind.print(dim);

    ind.set(
        dst, xyLdDst, ldldDst, ztLdDst,
        ind(a, xyLdA, ldldA, ztLdA) * ind(b, xyLdB, ldldB, ztLdB)
    );
}

