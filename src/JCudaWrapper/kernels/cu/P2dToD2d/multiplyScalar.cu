
#include <hip/hip_runtime.h>
/**
 * @class IndexMapper
 * @brief Maps a flattened thread index to 4D data indices (frame × depth × height × width),
 *        supporting column-major storage and strided memory layouts.
 */
class IndexMapper {
private:
    const int height;   ///< Height (rows) of each 2D layer.
    const int idx;     ///< Flattened thread/global index.
    const int sliceArea;     ///< Number of elements in a single 2D layer (height × width).
    const int depth;    ///< Index along the depth (z-axis) within a frame.
    const int frame;    ///< Index along the time (frame) axis.

public:
    /**
     * @brief Constructor that maps a flat index to 4D coordinates.
     *
     * @param globalIndex Global linear index across all 4D elements.
     * @param height      Height (rows) of each 2D layer.
     * @param width       Width (columns) of each 2D layer.
     * @param depth       Number of layers (z) per frame.
     */
    __device__ IndexMapper(const int globalIndex, const int* dim)
        : idx(globalIndex),
          height(dim[0]),
          sliceArea(dim[4]),
          depth((globalIndex / sliceArea) % dim[2]),
          frame(globalIndex / dim[5]) {}

    /**
     * @brief Multiplies the destination value by a scalar in-place.
     *
     * @param pointersToLayers 2D array of pointers to 2D layers (frames × depth).
     * @param ldLayers         2D array of column-major strides for each 2D layer.
     * @param ldld             Leading dimension of ldLayers (stride across depth).
     * @param ldPtrs           Leading dimension of pointersToLayers (stride across frames).
     * @param scalar           Scalar multiplier.
     */
    __device__ void multiply(double** pointersToLayers, const int* ldLayers, const int ldld, const int ldPtrs, double scalar) {
        pointersToLayers[layerPtrIndex(ldPtrs)][elementIndex(ldLayers, ldld)] *= scalar;
    }

    /**
     * @brief Computes column-major element index in the layer.
     *
     * @param ldLayers 2D array of per-layer column strides.
     * @param ldld     Leading dimension of ldLayers.
     * @return         Offset into the 2D layer data.
     */
    __device__ int elementIndex(const int* ldLayers, const int ldld) const {
        return (idx / height) * ldLayers[frame * ldld + depth] + (idx % height);
    }

    /**
     * @brief Computes the index into pointersToLayers.
     *
     * @param ldPtrs Leading dimension (stride) across frames in pointersToLayers.
     * @return       Index pointing to the correct 2D layer pointer.
     */
    __device__ int layerPtrIndex(const int ldPtrs) const {
        return frame * ldPtrs + depth;
    }
};


/**
 * @brief CUDA kernel that multiplies each element in a 4D dataset by a scalar.
 *
 * The dataset is organized as: frames × depth × height × width. Each 2D layer
 * is stored in column-major order and accessed via pointers with stride metadata.
 *
 * @param totalElements   Total number of elements to process.
 * @param pointersToLayers 2D array of pointers to 2D layers.
 * @param ldLayers        2D array of column strides per layer.
 * @param ldld            Leading dimension of ldLayers (stride across depth).
 * @param ldPtrs          Leading dimension of pointersToLayers (stride across frames).
 * @param height          Height (rows) of each 2D layer.
 * @param width           Width (columns) of each 2D layer.
 * @param depth           Depth (layers) per frame.
 * @param scalar          Scalar to multiply each element by.
 */
extern "C" __global__ void multiplyScalarKernel(
    const int totalElements,
    double** pointersToLayers, const int* ldLayers, const int ldld, const int ldPtrs,
    const int* dim,
    const double scalar
) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIndex >= totalElements) return;

    IndexMapper mapper(threadIndex, dim);
    mapper.multiply(pointersToLayers, ldLayers, ldld, ldPtrs, scalar);
}

