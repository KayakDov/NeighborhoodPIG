
#include <hip/hip_runtime.h>

extern "C" __global__
void atan2Kernel(const double* vectors, int ldFrom, double* angles, int incTo, int n) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;
    
    const double* vecFrom = vectors + idx*ldFrom;
    double* vecTo = angles + idx*incTo;
    
    *vecTo = atan2(vecFrom[1], vecFrom[0]);    
}
