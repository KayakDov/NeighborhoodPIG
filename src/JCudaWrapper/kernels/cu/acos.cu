
#include <hip/hip_runtime.h>

extern "C" __global__ void acosKernel(double *from, int incFrom, double *to, int incTo, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[idx * incTo] = acos(from[idx * incFrom]); 
    }
}
//nvcc -ptx acos.cu 
