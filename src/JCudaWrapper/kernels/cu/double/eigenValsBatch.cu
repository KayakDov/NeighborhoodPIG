#include <hip/hip_runtime.h>
#include <math.h>

class Val{
private:
    const int height;
    const int idx;
public:
    __device__ Val(int idx, int height): idx(idx), height(height){}
    /**
     * Retrieves a value from a column-major order matrix.
     *
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The value at the corresponding column-major index.
     */
    __device__ double get(const double* src, int ld) const{
	return src[(idx / height) * ld + (idx % height)];
    }
};

/**
 * Swap function for double values.
 *
 * @param a First value.
 * @param b Second value.
 */
__device__ inline void swap(double& a, double& b) {
    double temp = a;
    a = b;
    b = temp;
}

/**
 * Represents a 3x3 symmetric matrix in column-major format.
 */
class Matrix3x3 {
private:
    const double xx, xy, xz, yy, yz, zz;

public:
    /**
     * Constructs a Matrix3x3 object.
     *
     * @param xx Element at (0,0).
     * @param xy Element at (0,1) and (1,0).
     * @param xz Element at (0,2) and (2,0).
     * @param yy Element at (1,1).
     * @param yz Element at (1,2) and (2,1).
     * @param zz Element at (2,2).
     */
    __device__ explicit Matrix3x3(const double xx, const double xy, const double xz, const double yy, const double yz, const double zz) : xx(xx), xy(xy), xz(xz), yy(yy), yz(yz), zz(zz) {}

    /**
     * Computes the trace of the matrix.
     * @return The sum of the diagonal elements.
     */
    __device__ double trace() const {
        return xx + yy + zz;
    }

    /**
     * Computes the sum of 2x2 determinant minors of the matrix.
     * @return The sum of determinant minors.
     */
    __device__ double diagMinorSum() const {
        return yy*zz - yz*yz + xx*zz - xz*xz + xx*yy - xy*xy;
    }

    /**
     * Computes the determinant of the matrix.
     * @return The determinant value.
     */
    __device__ double determinant() const {
        return xx * (yy * zz - yz * yz) -
               xy * (xy * zz - yz * xz) +
               xz * (xy * yz - yy * xz);
    }
    
};

/**
 * Sorts a DIM-element array in descending order.
 * @param values Pointer to the array.
 */
__device__ static void sortDescending(double* values) {
    if(values[0] < values[1]) swap(values[0], values[1]);
    if(values[0] < values[2]) swap(values[0], values[2]);
    if(values[1] < values[2]) swap(values[1], values[2]);
}

/**
 * Represents an affine function y = ax + b.
 */
class Affine {
private:
    double a; /**< The slope of the line. */
    double b; /**< The y-intercept of the line. */

public:
    /**
     * Constructs an Affine function.
     * @param a The slope.
     * @param b The y-intercept.
     */
    __device__ Affine(double a, double b) : a(a), b(b) {}

    /**
     * Evaluates the function at a given x.
     * @param x The input value.
     * @return The corresponding y-value.
     */
    __device__ double operator()(double x) {
        return a * x + b;
    }

    /**
     * Maps multiple x-values to y-values.
     * @param x1 First x-value.
     * @param x2 Second x-value.
     * @param x3 Third x-value.
     * @param y Pointer to an array where results are stored.
     */
    __device__ void map(double x1, double x2, double x3, double* y) {
        y[0] = (*this)(x1);
        y[1] = (*this)(x2);
        y[2] = (*this)(x3);
    }
    
    /**
     * @return The slope of the function.
     */
    __device__ double getSlope(){
        return a;
    }
    
    /**
     * Prints the function parameters.
     */
    __device__ void print(){
        printf("a = %lf and b = %lf\n\n", a, b);
    }
};

/**
 * Computes the real roots of a cubic equation.
 *
 * @param b Coefficient of x^2.
 * @param c Coefficient of x.
 * @param d Constant term.
 * @param tolerance Numerical tolerance for zero checking.
 * @param val Output array to store roots.
 */
__device__ void cubicRoot(const double& b, const double& c, const double& d, const double tolerance, double* val){
    double p = (3*c - b*b)/9;
    double q = (2*b*b*b - 9*b*c + 27*d)/27;

    if (p > -tolerance) val[0] = val[1] = val[2] = -b / 3;
    else{
        Affine line(2 * sqrt(-p), -b/3);
    
        double inACos = q/(line.getSlope() * p);        
    
        if(inACos > 1 - 1e-10) line.map(1, -0.5, -0.5, val);
        else if(inACos < -1 + 1e-10) line.map(-1, 0.5, 0.5, val);
        else {
            double phi = acos(inACos);
            for(int i = 0; i < 3; i++) val[i] = line(cos((phi + i*2*M_PI)/3));
        }
    }
}

/**
 * CUDA Kernel to compute eigenvalues of a batch of 3x3 symmetric matrices.
 *
 * @param n Number of matrices.
 * @param srcHeight Height of the input matrices.
 * @param dst Pointer to the output eigenvalues.
 * @param ldDst Leading dimension of output.
 * @param tolerance Numerical tolerance for root computation.
 */
extern "C" __global__ void eigenValsBatchKernel(
    const int n, 
    const double* xx, const int ldxx, 
    const double* xy, const int ldxy, 
    const double* xz, const int ldxz,
    const double* yy, const int ldyy,
    const double* yz, const int ldyz,
    const double* zz, const int ldzz, 
    const int srcHeight, 
    double* dst, const int ldDst, int heightDst, 
    double tolerance
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n) return;
    
    Val src(idx, srcHeight);
    
    Matrix3x3 matrix(
    	src.get(xx, ldxx), 
    	src.get(xy, ldxy), 
    	src.get(xz, ldxz), 
    	src.get(yy, ldyy), 
    	src.get(yz, ldyz), 
    	src.get(zz, ldzz)
    );
    
    double* eigenvalues = dst + (3*idx/heightDst) * ldDst + (3 * idx) % heightDst;
    
    cubicRoot(-matrix.trace(), matrix.diagMinorSum(), -matrix.determinant(), tolerance, eigenvalues);
    
    sortDescending(eigenvalues);
}

