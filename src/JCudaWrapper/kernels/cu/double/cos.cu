
#include <hip/hip_runtime.h>

extern "C" __global__ void cosKernel(int n, double *from, int incFrom, double *to, int incTo) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[idx * incTo] = cos(from[idx * incFrom]); 
    }
}
//nvcc -ptx cos.cu 