
#include <hip/hip_runtime.h>

extern "C" __global__ void sqrtKernel(int n, double *from, int incFrom, double *to, int incTo) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[idx * incTo] = sqrt(from[idx * incFrom]); 
    }
}
//nvcc -ptx sqrt.cu 