
#include <hip/hip_runtime.h>
#include <cmath>  // For std::round
#include <hip/hip_math_constants.h>

/**
 * Computes the linear index in a column-major order matrix.
 *
 * @param idx The 1D index in a flattened array.
 * @param ld The leading dimension (stride between columns in memory).
 * @param height The number of rows in the matrix.
 * @return The column-major index.
 */
__device__ int ind(int idx, int ld, int height) {
    return (idx / height) * ld + (idx % height);
}

/**
 * @brief Computes the color intensity for a downward transition in the RGB spectrum.
 * 
 * @param a The base RGB component index (0-5) within the color cycle.
 * @param theta The angle in radians.
 * @return The calculated intensity value (0-255) for the downward transition.
 */
__device__ double down(int a, double theta) {
    return 255 * (1 + a - 3 * theta / HIP_PI);
}

/**
 * @brief Computes the color intensity for an upward transition in the RGB spectrum.
 * 
 * @param a The base RGB component index (0-5) within the color cycle.
 * @param theta The angle in radians.
 * @return The calculated intensity value (0-255) for the upward transition.
 */
__device__ double up(int a, double theta) {
    return 255 * (3 * theta / HIP_PI - a);
}

/**
 * @class Writer
 * @brief A helper class that writes RGB values to the output array.
 */
class Writer {
private:    
    int* writeTo;     ///< Pointer to the output location in the colors array.
    double intensity; ///< Intensity scaling factor.

public:
    /**
     * @brief Constructs a Writer object to store RGB values.
     * 
     * @param writeTo Pointer to the output location in the colors array.
     * @param intensity Scaling factor for intensity (1.0 if no scaling is applied).
     */
    __device__ Writer(int* writeTo, double intensity)
        : writeTo(writeTo), intensity(intensity) {}

    /**
     * @brief Rounds the value and multiplies it by intensity.
     *
     * @param c The input color component.
     * @return The rounded intensity-scaled value.
     */
    __device__ int roundValue(double c) {
        return static_cast<int>(rint(c * intensity));
    }

    /**
     * @brief Stores an RGB color value, applying intensity scaling.
     * 
     * @param r Red component (0-255).
     * @param g Green component (0-255).
     * @param b Blue component (0-255).
     */
    __device__ void setColor(double r, double g, double b) {
        *writeTo = (roundValue(r) << 16) | (roundValue(g) << 8) | roundValue(b);
    }
};

/**
 * @brief CUDA kernel to compute RGB color values from input angles and intensities.
 * 
 * @details This kernel processes n input angles, converting each to an RGB color based on its position in the spectrum.
 *          The results are stored in column-major order. Optionally, intensity scaling is applied.
 * 
 * @param n The number of input angles to process.
 * @param srcAngles Pointer to the column-major input array of angles (in radians).
 * @param ldSrcAng Leading dimension (stride) of srcAngles in memory.
 * @param heightSrcAng Number of rows in srcAngles (defines how many values belong to one column).
 * @param colors Pointer to the column-major output array for storing RGB values.
 * @param ldCol Leading dimension (stride) of colors in memory.
 * @param heightCol Number of rows in colors (defines how many values belong to one column).
 * @param srcIntensities Pointer to the column-major array of intensity values (optional).
 * @param heightSrcInt Number of rows in srcIntensities. Ignored if srcIntensities == nullptr.
 * @param ldSrcInt Leading dimension (stride) of srcIntensities. Pass -1 if srcIntensities is unused.
 */
extern "C" __global__ void colorsKernel(
    const int n, 
    
    const double* srcAngles, 
    const int ldSrcAng, 
    const int heightSrcAng,
    
    int* colors,
    const int ldCol,
    const int heightCol,
    
    const double* srcIntensities,
    const int heightSrcInt,
    const int ldSrcInt    
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    double angle = srcAngles[ind(idx, ldSrcAng, heightSrcAng)];    
    
    double intensity = (ldSrcInt == -1) ? 1.0 : srcIntensities[ind(idx, ldSrcInt, heightSrcInt)];
    
    Writer writer(colors + ind(idx, ldCol, heightCol), intensity);
    
    
    if(isnan(angle)) writer.setColor(0, 0, 0);
	else if (-1e-5 <= angle && angle < HIP_PI / 3) 
        writer.setColor(255, up(0, angle), 0);
    
    else if (HIP_PI / 3 <= angle && angle < 2 * HIP_PI / 3) 
        writer.setColor(down(1, angle), 255, 0);
    
    else if (2 * HIP_PI / 3 <= angle && angle < HIP_PI) 
        writer.setColor(0, 255, up(2, angle));
    
    else if (HIP_PI <= angle && angle < 4 * HIP_PI / 3) 
        writer.setColor(0, down(3, angle), 255);
    
    else if (4 * HIP_PI / 3 <= angle && angle < 5 * HIP_PI / 3) 
        writer.setColor(up(4, angle), 0, 255);
    
    else if (5 * HIP_PI / 3 <= angle && angle <= 2 * HIP_PI + 1e-5) 
        writer.setColor(255, 0, down(5, angle));
    else writer.setColor(0, 0, 0);
}
