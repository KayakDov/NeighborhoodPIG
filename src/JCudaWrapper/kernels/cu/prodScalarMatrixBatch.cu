
#include <hip/hip_runtime.h>

//please ensure that n = batchSize * height * width
extern "C" __global__ void prodScalarMatrixBatchKernel(double *scalars, int inc, double *to, int height, int n, int width, int colDist, int stride) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int matrixSize = height*width;
    
    if (idx < n) {
    
    	int fromIndex = idx / matrixSize;
    	int inMatrixIndex = idx % matrixSize;
    	
    	int colIndex = inMatrixIndex / height;
    	int rowIndex = inMatrixIndex % height;
        
        int toIndex = fromIndex * stride + colIndex * colDist + rowIndex;
        
        to[toIndex] *= scalars[fromIndex*inc]; 
    }
}
//nvcc -ptx fillMatrix.cu 
