#include <hip/hip_runtime.h>
#include <math.h>


/**
 * Uses Kahan's method for more accurate mulitplication.
 */
__device__ double prod(double a, double b){
    double result = a*b;
    return result - fma(a, b, -result);
}

/**
 * Swap function for double values.
 *
 * @param a First value.
 * @param b Second value.
 */
__device__ inline void swap(float& a, float& b) {
    float temp = a;
    a = b;
    b = temp;
}
/**
 * Swap function for double values.
 *
 * @param a First value.
 * @param b Second value.
 */
__device__ inline void swap(double& a, double& b) {
    double temp = a;
    a = b;
    b = temp;
}

/**
 * @class Get
 * @brief A helper class for accessing values in a column-major order matrix.
 */
class Get{
private:
    const int height;
    const int idx;
    const int downSampleFactorXY;
public:
    __device__ Get(const int inputIdx, const int height, const int downSampleFactorXY)
: idx(inputIdx * downSampleFactorXY), height(height), downSampleFactorXY(downSampleFactorXY) {}

    
    /**
     * @brief Retrieves a value from a column-major order matrix.
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The value at the corresponding column-major index.
     */
    __device__ float val(const float* src, const int  ld) const{
		return val(src, height, ld);		
    }
    
    /**
     * @brief Retrieves a value from a column-major order matrix.
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @param height The height of the matrix.
     * @return The value at the corresponding column-major index.
     */
    __device__ float val(const float* src, const int height, const int ld) const{
		return src[ind(height, ld)];
    }
    
    /**
     * @brief Retrieves an index from a column-major order matrix.
     * @param height The height of the matrix.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The computed column-major index.
     */
    __device__ int ind(const int height, const int ld) const{
		return downSampleFactorXY * (idx / height) * ld + idx % height;
    }
    
    /**
     * @brief Retrieves an index from a column-major order matrix using stored height.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The computed column-major index.
     */
    __device__ int ind(const int ld) const{
		return ind(height, ld);
    }
    
    
};

/**
 * @class MaxAbs
 * @brief A utility class for tracking the argument corresponding to the maximum absolute value in a set of comparisons.
 *
 * This class is designed for use in CUDA device code and provides methods to update the tracked maximum 
 * absolute value and retrieve the corresponding argument.
 */
class MaxAbs {
private:
    int arg; ///< The argument corresponding to the maximum absolute value.
    double val; ///< The maximum absolute value encountered so far.    

public:
    /**
     * @brief Constructor for the MaxAbs class.
     * 
     * Initializes the maximum absolute value and its corresponding argument.
     *
     * @param initVal The initial maximum absolute value.
     * @param initArg The initial argument corresponding to the maximum absolute value.
     */
    __device__ MaxAbs(int initArg, double initVal) : arg(initArg), val(fabs(initVal)) {}

     /**
     * @brief Updates the tracked maximum absolute value if the new value is greater.
     * 
     * Compares the given value with the current maximum absolute value. If the new value is greater,
     * updates the maximum value and its corresponding index.
     *
     * @param candidateIndex The index associated with the new value.
     * @param candidateValue The new value to compare against the current maximum absolute value.
     */
    __device__ void challenge(int candidateIndex, double candidateValue) {
        double absoluteValue = fabs(candidateValue); // Compute the absolute value of the candidate value.
        if (absoluteValue > val) {             // Update if the candidate value is larger than the current maximum.
            val = absoluteValue;
            arg = candidateIndex;
        }
    }

    /**
     * @brief Retrieves the argument corresponding to the maximum absolute value.
     *
     * @return The argument corresponding to the maximum absolute value.
     */
    __device__ int getArg() {
        return arg;
    }
    
    /**
     * @brief Retrieves the absolute value at the argument.
     *
     * @return The maximum absolute value.
     */
    __device__ double getVal() {
        return val;
    }
};

/**
 * Represents a 3x3 symmetric matrix in column-major format.
 */
class Matrix3x3 {
private:
    double mat[3][3];
    double tolerance;
    
    /**
     * A value that is less than tolerance will be returned as 0.  Otherwise as itself.
     */
    __device__ double zeroBar(double maybeNear0){
        return fabs(maybeNear0) <= tolerance? 0: maybeNear0;
    }
public:
    /**
     * Constructs a Matrix3x3 object.
     *
     * @param xx Element at (0,0).
     * @param xy Element at (0,1) and (1,0).
     * @param xz Element at (0,2) and (2,0).
     * @param yy Element at (1,1).
     * @param yz Element at (1,2) and (2,1).
     * @param zz Element at (2,2).
     * @param tol the tolerance.
     */
    __device__ explicit Matrix3x3(const float xx, const float xy, const float xz, const float yy, const float yz, const float zz, double tol) : tolerance(tol) {
        mat[0][0] = zeroBar(xx);
        mat[0][1] = mat[1][0] = zeroBar(xy);
        mat[0][2] = mat[2][0] = zeroBar(xz);
        mat[1][1] = zeroBar(yy);
        mat[1][2] = mat[2][1] = zeroBar(yz);
        mat[2][2] = zeroBar(zz);
    }



    /**
     * Computes the trace of the matrix.
     * @return The sum of the diagonal elements.
     */
    __device__ double trace() const {
        return mat[0][0] + mat[1][1] + mat[2][2];
    }

    /**
     * Computes the sum of 2x2 determinant minors of the matrix.
     * @return The sum of determinant minors.
     */
    __device__ double diagMinorSum() const {
        return mat[1][1]*mat[2][2] - mat[1][2]*mat[1][2] + mat[0][0]*mat[2][2] - mat[0][2]*mat[0][2] + mat[0][0]*mat[1][1] - mat[0][1]*mat[0][1];
    }

    /**
     * Computes the determinant of the matrix.
     * @return The determinant value.
     */
    __device__ double determinant() const {
        return mat[0][0] * (mat[1][1] * mat[2][2] - mat[1][2] * mat[1][2]) -
               mat[0][1] * (mat[0][1] * mat[2][2] - mat[1][2] * mat[0][2]) +
               mat[0][2] * (mat[0][1] * mat[1][2] - mat[1][1] * mat[0][2]);
    }
    
    /**
     * Subtracts the val from each element on the diagnal of this matrix, changing this matrix.
     * @param val The value to be subtracted from each element of this matrix.
     */
    __device__ void subtractFromDiag(double val){
        mat[0][0] -= val; mat[1][1] -= val; mat[2][2] -= val;
    }
    
    /**
     * @brief Access an element in the matrix by row and column index.
     * @param row Row index.
     * @param col Column index.
     * @return Reference to the element at the specified row and column.
     */
    __device__ double& operator()(int row, int col) {
        return mat[row][col];
    }
    
    /**
     * @brief Subtracts a scaled row from another row.
     * @param minuendInd Index of the row to be updated.
     * @param subtrahendInd Index of the row to subtract.
     * @param scale Scaling factor.
     * @param The value at this column will be set to 0.  Perform subtraction to the right.
     */
    __device__ void subtractRow(int minuendInd, int subtrahendInd, double scale, int startCol) {
        mat[minuendInd][startCol] = 0;
        for (int i = startCol + 1; i < 3; i++){
            if(fabs(mat[minuendInd][i]) <= tolerance && fabs(mat[subtrahendInd][i]) <= tolerance) mat[minuendInd][i] = 0;
            else if(fabs(mat[minuendInd][i]) <= tolerance) mat[minuendInd][i] = prod(scale, mat[subtrahendInd][i]);
            else if(fabs(mat[subtrahendInd][i]) <= tolerance) mat[minuendInd][i] = -mat[minuendInd][i];
            else 
                mat[minuendInd][i] = fma(scale, mat[subtrahendInd][i], -mat[minuendInd][i]);
        }
        
    }

    /**
     * @brief Swaps two rows of the matrix.
     * @param i First row index.
     * @param j Second row index.
     * @param startCol begin swaping with this column and proceed to the right.
     */
    __device__ void swapRows(int i, int j, int startCol) {
        for(int k = startCol; k < 3; k++) swap(mat[i][k], mat[j][k]);
    }
    
    /**
     * Scales the row so that the element at the startCol is one and every element after is times one over that element.
     * @param row the row to be scaled.
     * @startCol the column index of the first non zero element of the row.
     */
    __device__ void scaleRow(int row, int startCol){
	
	double inv = 1/mat[row][startCol]; 
    	mat[row][startCol] = 1;
    	for(int i = startCol + 1; i < 3; i++) mat[row][i] *= inv;
    	
    }

    
    /**
     * @brief Perform row echelon work for a specific row and column.
     * @param row Current row index.
     * @param col Current column index.
     * @return True if a pivot was found, false otherwise.
     */
    __device__ bool reduceToRowEchelon(const int row, const int col) {
        
        MaxAbs maxPivot(row, fabs(mat[row][col]));
        
	for (int i = row + 1; i < 3; i++) maxPivot.challenge(i, mat[i][col]);
	

        if (maxPivot.getVal() <= tolerance) return false;

        if (maxPivot.getArg() != row) swapRows(maxPivot.getArg(), row, col);
        
        for (int i = row + 1; i < 3; i++){
	    subtractRow(i, row, mat[i][col]/mat[row][col], col);
	    scaleRow(row, col);
	}

        return true;
    }

    /**
     * @brief Perform row echelon reduction on the matrix.
     * @return Number of free variables found during the reduction.
     */
    __device__ int rowEchelon() {
        int numFreeVariables = 0;
        int row = 0;

        for (int col = 0; col < 3; col++)           
            if (reduceToRowEchelon(row, col)) row++;
            else numFreeVariables++;
            
        return numFreeVariables;
    }
    
    /**
     * Prints the matrix for debugging purposes using a single printf.
     */
    __device__ void print() {
        printf("\nMatrix:\n%f %f %f\n%f %f %f\n%f %f %f\n",
               mat[0][0], mat[0][1], mat[0][2],
               mat[1][0], mat[1][1], mat[1][2],
               mat[2][0], mat[2][1], mat[2][2]);
    }
    
};

/**
 * Represents an affine function y = ax + b.
 */
class Affine {
private:
    double a; /**< The slope of the line. */
    double b; /**< The y-intercept of the line. */

public:
    /**
     * Constructs an Affine function.
     * @param a The slope.
     * @param b The y-intercept.
     */
    __device__ Affine(double a, double b) : a(a), b(b) {}

    /**
     * Evaluates the function at a given x.
     * @param x The input value.
     * @return The corresponding y-value.
     */
    __device__ double operator()(double x) {
        return fma(a, x, b);
    }

    /**
     * Maps multiple x-values to y-values.
     * @param x1 First x-value.
     * @param x2 Second and thid x-values.
     * @param y Pointer to an array where results are stored.
     */
    __device__ void map(double x1, double x2And3, double* y) {
        y[0] = (*this)(x1);
        y[1] = y[2] = (*this)(x2And3);
    }
    
    /**
     * @return The slope of the function.
     */
    __device__ double getSlope(){
        return a;
    }
    
    /**
     * Prints the function parameters.
     */
    __device__ void print(){
        printf("a = %lf and b = %lf\n\n", a, b);
    }
};


/**
 * @class Vec
 * @brief A simple wrapper for a double array representing a 3D vector.
 */
class Vec {
private:
    float* data;
public:
    /**
     * @brief Constructs a Vec object.
     * @param data Pointer to the float array (size 3) representing the vector.
     */
    __device__ Vec(float* data):data(data){}

    /**
     * @brief Sets the components of the vector.
     * @param x The x-component.
     * @param y The y-component.
     * @param z The z-component.
     */
    __device__ void set(float x, float y, float z){
        data[0] = x; data[1] = y; data[2] = z;
    }

    /**
     * @brief Accesses a component of the vector using array-like indexing.
     * @param i The index of the component (0 for x, 1 for y, 2 for z).
     * @return A reference to the requested vector component.
     */
    __device__ float& operator[](int i) {
        return data[i];
    }
    
    /**
     * @brief Prints the components of the vector to the standard output.
     * The output format is "(x, y, z)".
     */
    __device__ void print() const {
        printf("(%f, %f, %f)\n", data[0], data[1], data[2]);
    }
    
    /**
     * @brief Checks if any of the vector's components are NaN (Not a Number).
     * @return True if at least one component is NaN, false otherwise.
     */
    __device__ bool hasNaN() const {
        return isnan(data[0]) || isnan(data[1]) || isnan(data[2]);
    }
    
        /**
     * @brief Calculates the squared length (magnitude) of the vector.
     * @return The squared length of the vector.
     */
    __device__ double lengthSquared() const {
        return data[0] * data[0] + data[1] * data[1] + data[2] * data[2];
    }

    /**
     * @brief Calculates the length (magnitude) of the vector.
     * @return The length of the vector.
     */
    __device__ double length() const {
        return sqrtf(lengthSquared());
    }

    /**
     * @brief Normalizes the vector in-place, setting its length to 1.
     * If the vector's length is zero, it remains unchanged.
     */
    __device__ void normalize() {
        double len = length();
        if (len > 0.0f) {
            double invLen = 1.0 / len;
            data[0] *= invLen;
            data[1] *= invLen;
            data[2] *= invLen;
        }
    }
    
};

class EVal{
private:
    double data[3];
    
    /**
     * Sorts an array in descending order.
     */
    __device__ void sortDescending() {
        if(data[0] < data[1]) swap(data[0], data[1]);
        if(data[0] < data[2]) swap(data[0], data[2]);
        if(data[1] < data[2]) swap(data[1], data[2]);
    }
    
    
    /**
     * @brief Sets the components of the vector.
     * @param x The x-component.
     * @param y The y-component.
     * @param z The z-component.
     */
    __device__ void set(double x, double y, double z){
        data[0] = x; data[1] = y; data[2] = z;
    }
    
    /**
     * Writes these values to the desired location.
     */
    __device__ void writeTo(float* to){
        to[0] = data[0]; to[1] = data[1]; to[2] = data[2];
    }

    /**
     * Computes the real roots of a cubic equation.
     *
     * @param b Coefficient of x^2.
     * @param c Coefficient of x.
     * @param d Constant term.
     * @param eigenInd The index of the eigenvalue to be returned from this method.  0 for the largest eigenValue and 2 for the smallest.
     * @param val Output array to store roots.
     * @return The eigen value at the desired index.
     */
    __device__ void cubicRoot(const double b, const double c, const double d){
	
	double inv3 = 1.0/3;
	
	double nBInv3 = -b*inv3;
	
	double p = fma(nBInv3, b, c) * inv3;
	double q = fma(fma(b/13.5, b, -c*inv3), b, d);

	if (p >= -1e-9) set(nBInv3, nBInv3, nBInv3);
	
	else{
	    
	    Affine line(2 * sqrt(-p), nBInv3);
	
	    double arg = q/prod(line.getSlope(), p);
	
	    if(arg > 1 - 1e-6) line.map(1, -0.5, data);
	    else if(arg < -1 + 1e-6) line.map(-1, 0.5, data);
	    else {

	        double acosArg = acos(arg); 

	        set(line(cos(acosArg * inv3)), 
 	            line(cos(fma(2, M_PI, acosArg) * inv3)), 
	            line(cos(fma(4, M_PI, acosArg) * inv3))
	    	);
	    }
	    		   
	   if(blockIdx.x * blockDim.x + threadIdx.x == 575*1153 + 150){
	       printf("eigenBatch Has eigenvalues (%lf, %lf, %lf)\n", data[0], data[1], data[2]);
	   }		   
	}
    }
public:

    /**
     * Finds the eigenvalues.
     *@param mat The matrix for whom the eigenvalues are desired.
     */
    __device__ EVal(const Matrix3x3& mat, float* dst){
       
 
        cubicRoot(-mat.trace(), mat.diagMinorSum(), -mat.determinant());
        sortDescending();
        writeTo(dst);
    }
    
    __device__ int multiplicity(int ind){
        return (data[0] == data[ind]) + (data[1] == data[ind]) + (data[2] == data[ind]) - 1;
    }
    
    
    /**
     * @brief Accesses a component of the vector using array-like indexing.
     * @param i The index of the component (0 for x, 1 for y, 2 for z).
     * @return A reference to the requested vector component.
     */
    __device__ double& operator[](int i) {
        return data[i];
    }
};


/**
 * CUDA Kernel to compute eigenvalues of a batch of 3x3 symmetric matrices.
 *
 * @param n Number of matrices fordownSampleFactorXY = 1, even if it's not.
 * @param srcHeight Height of the input matrices.
 * @param valDst Pointer to the output eigenvalues.
 * @param ldEVal Leading dimension of output.
 * @param 1 of every how many structure tensors should be evaluated in the x and y dimensions.
 */
extern "C" __global__ void eigenBatchKernel(
    const int n, 
    
    const float* xx, const int ldxx, 
    const float* xy, const int ldxy, 
    const float* xz, const int ldxz,
    const float* yy, const int ldyy,
    const float* yz, const int ldyz,
    const float* zz, const int ldzz, 
    
    const int srcHeight, 
    
    float* valDst, const int ldEVal, int heightValDst, 
    
    const int downSampleFactorXY, const int eigenInd,
    
    float* vecDst, const int ldEVec, const int heightEVec,
    
    double tolerance
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n/downSampleFactorXY/downSampleFactorXY) return;
    
    Get src(idx, srcHeight, downSampleFactorXY);
    
    Matrix3x3 mat(
    	src.val(xx, ldxx), src.val(xy, ldxy), src.val(xz, ldxz), 
                           src.val(yy, ldyy), src.val(yz, ldyz), 
    					      src.val(zz, ldzz),
        tolerance
    );
    
    Get getx3(3*idx, heightEVec, 1);
    
    EVal eVals(mat, valDst + getx3.ind(ldEVal));
    
    mat.subtractFromDiag(eVals[eigenInd]);
    
    if(idx == 575*srcHeight + 150) mat.print();
    
    Vec vec(vecDst + getx3.ind(ldEVec));
    
    int freeVariables = mat.rowEchelon();
    
    double smTol = 1e-8;
    
    switch(freeVariables){
    
	case 1:
	    if(fabs(mat(0, 0)) <= smTol) vec.set(1, 0, 0);
	    else if(fabs(mat(1, 1)) <= smTol) vec.set(-mat(0,1)/mat(0,0), 1, 0);
	    else {
	        vec[2] = 1; 
	        vec[1] = -mat(1,2)/mat(1,1); 
	        vec[0] = (-mat(0,2) - mat(0,1)*vec[1])/mat(0,0);
	    }
	    break;
	    
	case 2:
	    if(fabs(mat(0,0)) <= smTol)
	        if(fabs(mat(0, 1)) <= smTol)
	            if(eigenInd % 2 == 0) vec.set(1, 0, 0);
	            else vec.set(0, 1, 0);
	        else if(eigenInd % 2 == 0) vec.set(1, 0, 0);
	            else vec.set(0, -mat(0, 2)/mat(0, 1), 1);
	    else {
	    	switch(eigenInd){
	    	    case 0:
	    	        if(fabs(mat(0, 1)) >= smTol) vec.set(0, -mat(0, 2)/mat(0, 1), 1);
	    	    	else vec.set(0, 1, 0);
	    	    	break;
	    	    case 1:  vec.set(-mat(0, 1)/mat(0, 0), 1, 0); break;	    	    
                    case 2: vec.set(-mat(0, 2)/mat(0, 0), 0, 1); 
	    	}
	    }
	    break;
	    
	case 3:
	    switch(eigenInd){
	        case 0: vec.set(1, 0, 0); break;
	        case 1: vec.set(0, 1, 0); break;
	        case 2: vec.set(0, 0, 1);
	    }	    
    }
    
    if(idx == 575*srcHeight + 150) {
        printf("vec in eigenVecBatch3x3 with index %d -> (%d, %d) is : (%f. %f, %f)\nwith tolerance %f\nAnd free variables %d\nAnd eigenInd = %d\n\n", idx, idx/srcHeight, idx%srcHeight, vec[0], vec[1], vec[2], tolerance, freeVariables, eigenInd);
        mat.print();    
    }

    
//    if(idx == 0) {mat.print(); vec.print();}

    vec.normalize();
}

