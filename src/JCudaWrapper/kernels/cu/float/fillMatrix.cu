
#include <hip/hip_runtime.h>
//The first value of from us used to fill this matrix.
//n is the size of the matrix, width * height.

extern "C" __global__ void fillMatrixKernel(int n, float *from, int lda, float *to, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[(idx/height)*lda + idx % height] = from[0]; 
    }
}
//nvcc -ptx fillMatrix.cu 