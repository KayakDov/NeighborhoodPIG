#include <hip/hip_runtime.h>
#include <math.h>

class Val{
private:
    const int height;
    const int idx;
    const int downSampleFactorXY;
public:
    __device__ Val(const int idx, const int height, const int downSampleFactorXY): idx(idx*downSampleFactorXY), height(height), downSampleFactorXY(downSampleFactorXY){}
    /**
     * Retrieves a value from a column-major order matrix.
     *
     * @param src Pointer to the source array.
     * @param ld The leading dimension (stride between columns in memory).
     * @return The value at the corresponding column-major index.
     */
    __device__ float get(const float* src, const int ld) const{
	return src[downSampleFactorXY * (idx / height) * ld + idx % height];
    }
};

/**
 * Swap function for double values.
 *
 * @param a First value.
 * @param b Second value.
 */
__device__ inline void swap(float& a, float& b) {
    float temp = a;
    a = b;
    b = temp;
}

/**
 * Represents a 3x3 symmetric matrix in column-major format.
 */
class Matrix3x3 {
private:
    const double xx, xy, xz, yy, yz, zz;

public:
    /**
     * Constructs a Matrix3x3 object.
     *
     * @param xx Element at (0,0).
     * @param xy Element at (0,1) and (1,0).
     * @param xz Element at (0,2) and (2,0).
     * @param yy Element at (1,1).
     * @param yz Element at (1,2) and (2,1).
     * @param zz Element at (2,2).
     */
    __device__ explicit Matrix3x3(const float xx, const float xy, const float xz, const float yy, const float yz, const float zz) : xx(xx), xy(xy), xz(xz), yy(yy), yz(yz), zz(zz) {}

    /**
     * Computes the trace of the matrix.
     * @return The sum of the diagonal elements.
     */
    __device__ double trace() const {
        return xx + yy + zz;
    }

    /**
     * Computes the sum of 2x2 determinant minors of the matrix.
     * @return The sum of determinant minors.
     */
    __device__ double diagMinorSum() const {
        return yy*zz - yz*yz + xx*zz - xz*xz + xx*yy - xy*xy;
    }

    /**
     * Computes the determinant of the matrix.
     * @return The determinant value.
     */
    __device__ double determinant() const {
        return xx * (yy * zz - yz * yz) -
               xy * (xy * zz - yz * xz) +
               xz * (xy * yz - yy * xz);
    }
    
};

/**
 * Sorts a DIM-element array in descending order.
 * @param values Pointer to the array.
 */
__device__ static void sortDescending(float* values) {
    if(values[0] < values[1]) swap(values[0], values[1]);
    if(values[0] < values[2]) swap(values[0], values[2]);
    if(values[1] < values[2]) swap(values[1], values[2]);
}

/**
 * Represents an affine function y = ax + b.
 */
class Affine {
private:
    double a; /**< The slope of the line. */
    double b; /**< The y-intercept of the line. */

public:
    /**
     * Constructs an Affine function.
     * @param a The slope.
     * @param b The y-intercept.
     */
    __device__ Affine(double a, double b) : a(a), b(b) {}

    /**
     * Evaluates the function at a given x.
     * @param x The input value.
     * @return The corresponding y-value.
     */
    __device__ double operator()(double x) {
        return a * x + b;
    }

    /**
     * Maps multiple x-values to y-values.
     * @param x1 First x-value.
     * @param x2 Second x-value.
     * @param x3 Third x-value.
     * @param y Pointer to an array where results are stored.
     */
    __device__ void map(double x1, double x2, double x3, float* y) {
        y[0] = (*this)(x1);
        y[1] = (*this)(x2);
        y[2] = (*this)(x3);
    }
    
    /**
     * @return The slope of the function.
     */
    __device__ double getSlope(){
        return a;
    }
    
    /**
     * Prints the function parameters.
     */
    __device__ void print(){
        printf("a = %lf and b = %lf\n\n", a, b);
    }
};

/**
 * Computes the real roots of a cubic equation.
 *
 * @param b Coefficient of x^2.
 * @param c Coefficient of x.
 * @param d Constant term.
 * @param val Output array to store roots.
 */
__device__ void cubicRoot(const double b, const double c, const double d, float* val){
    double bSq = b*b;
    double p = c/3 - bSq/9;
    double q = b*bSq/13.5 - b*c/3 + d;

    if (p >= -1e-9) val[0] = val[1] = val[2] = -b / 3;
    else{
        Affine line(2 * sqrt(-p), -b/3);
    
        double inACos = q/(line.getSlope() * p);        
    
        if(inACos > 1 - 1e-6) line.map(1, -0.5, -0.5, val);
        else if(inACos < -1 + 1e-6) line.map(-1, 0.5, 0.5, val);
        else for(int i = 0; i < 3; i++) val[i] = line(cos((acos(inACos) + i*M_PI*2)/3));        
    }
}

/**
 * CUDA Kernel to compute eigenvalues of a batch of 3x3 symmetric matrices.
 *
 * @param n Number of matrices fordownSampleFactorXY = 1, even if it's not.
 * @param srcHeight Height of the input matrices.
 * @param dst Pointer to the output eigenvalues.
 * @param ldDst Leading dimension of output.
 * @param 1 of every how many structure tensors should be evaluated in the x and y dimensions.
 */
extern "C" __global__ void eigenValsBatchKernel(
    const int n, 
    const float* xx, const int ldxx, 
    const float* xy, const int ldxy, 
    const float* xz, const int ldxz,
    const float* yy, const int ldyy,
    const float* yz, const int ldyz,
    const float* zz, const int ldzz, 
    const int srcHeight, 
    float* dst, const int ldDst, int heightDst, 
    const int downSampleFactorXY
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n/downSampleFactorXY/downSampleFactorXY) return;
    
    Val src(idx, srcHeight, downSampleFactorXY);
    
    Matrix3x3 matrix(
    	src.get(xx, ldxx), src.get(xy, ldxy), src.get(xz, ldxz), 
                           src.get(yy, ldyy), src.get(yz, ldyz), 
    					      src.get(zz, ldzz)
    );
    
    float* eigenvalues = dst + (3*idx/heightDst) * ldDst + (3 * idx) % heightDst;
    
    cubicRoot(-matrix.trace(), matrix.diagMinorSum(), -matrix.determinant(), eigenvalues);
    
    sortDescending(eigenvalues);
}

