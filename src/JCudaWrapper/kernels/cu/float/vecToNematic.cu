
#include <hip/hip_runtime.h>
__device__ int ind(int useableInd, int ld, int height){
	return 3 * useableInd / height * ld + 3 * useableInd % height;
}

//Takes a regular 3d vector with angle between 0 and 2pi and maps it to a nematic vector between 0 and pi.

extern "C" __global__
void vecToNematicKernel(int n, const float* src, const int ldSrc, const int heightSrc, float* dst, const int ldDst, const int heightDst) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n) return;
    
    const float* vec = src + ind(idx, ldSrc, heightSrc);
    float* nematic = dst + ind(idx, ldDst, heightDst);

    if(vec[1] < 0 || (vec[1] == 0 && vec[0] < 0)) for(int i = 0; i < 3; i++) nematic[i] = -1 * vec[i];
    else if(dst != src) for(int i = 0; i < 3; i++) nematic[i] = vec[i];
        
}
