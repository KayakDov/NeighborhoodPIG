
#include <hip/hip_runtime.h>

extern "C" __global__ void cosKernel(int n, float *from, int incFrom, float *to, int incTo) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        to[idx * incTo] = cos(from[idx * incFrom]); 
    }
}
//nvcc -ptx cos.cu 